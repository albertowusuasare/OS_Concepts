#include "hip/hip_runtime.h"
/* A parallel program using CUDA to invert a PBM image
 *
 * Created by Jerod Weinman, 2 November 2012
 * Revised 13 August 2014
 */

#include "cubitmap.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define TILE_WIDTH 4

/* Kernel that performs inversion on a subset of pixels 
 * img is the original image on the device 
 */
__global__ void invertKernel(bit *img, int rows, int cols)
{

  int i = blockIdx.y * TILE_WIDTH + threadIdx.y; /* Pixel row */
  int j = blockIdx.x * TILE_WIDTH + threadIdx.x; /* Pixel col */
  int index = i*cols + j;   /* Pixel linear index */
  
  if (i<rows && j<cols)
    img[index] = !img[index];   /* Flip the bit */
}

/* Host function to invert an image */  
__host__ int invert( bm_t *p_orig_h )
{

  int rows = p_orig_h->rows;
  int cols = p_orig_h->cols;

  /* Allocate space on device */
  bit* p_orig_d = cu_bmalloc(rows, cols);
  bit* p_rest_d = cu_bmalloc(rows, cols);
  bit* p_work_d = cu_bmalloc(rows, cols);

  if (p_orig_d==NULL || p_rest_d==NULL || p_work_d==NULL) 
  {
    fprintf(stderr,"Error allocating device image memory\n");
    exit(EXIT_FAILURE);
  }
  
  if ( hd_bmcopy( p_orig_d, p_orig_h) < 0 ) /* Copy image to device */
  {
    fprintf(stderr,"Error copying original image to device\n");
    exit(EXIT_FAILURE);
  }


  /* Set up execution configuration */
  dim3 dimGrid( cols / TILE_WIDTH + 1, rows / TILE_WIDTH + 1 );
  dim3 dimBlock( TILE_WIDTH, TILE_WIDTH );
  
  invertKernel<<<dimGrid,dimBlock>>>(p_orig_d, rows, cols ); /* Invoke */

  if (dh_bmcopy( p_orig_h, p_orig_d)<0) /* Copy inverted image to host */
  {
    fprintf(stderr,"Error copying inverted image to host");
    exit(EXIT_FAILURE);
  }

  if (hipFree( p_orig_d ) != hipSuccess ||   /* Free device memory */
      hipFree( p_rest_d ) != hipSuccess ||
      hipFree( p_work_d )  != hipSuccess ) {
    fprintf(stderr,"Error releasing device memory\n");
  }
  return 0;
}

/* Usage: cuinvert input output */
int main(int argc, char* argv[])
{
  char *origFile, *invertedFile;
  bm_t image;
  int result;
  struct timeval start,end,diff;
  
  /* Process command line arguments */
  if (argc!=3)
  {
    fprintf(stderr,"Usage: %s input output\n", argv[0]);
    exit(EXIT_FAILURE);
  }
  
  origFile = argv[1];
  invertedFile = argv[2];

  if ( bmread(origFile, &image) < 0 )   /* Read input image */
    exit(EXIT_FAILURE);
  
  result = gettimeofday(&start, NULL);  /* Start timer */
  
  if (result) 
  {
    perror("Could not get start time");
    exit(EXIT_FAILURE);
  }
  
  invert(&image);                      /* Run invert */

  result = gettimeofday(&end, NULL);   /* End timer */
  
  if (result) 
  {
    perror("Could not get end time");
    exit(EXIT_FAILURE);
  }
  
  timersub(&end, &start, &diff);       /* Calculate interval */

  printf("%3u.%06u\n",diff.tv_sec,diff.tv_usec);

  if ( bmwrite(invertedFile, &image) < 0 )
    exit (EXIT_FAILURE);
  
  exit (EXIT_SUCCESS);
}
