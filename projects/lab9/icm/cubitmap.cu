/* cubitmap
 * 
 * Routines for allocating, freeing, and moving the bitmap image
 * format to and from CUDA devices
 *
 * Created by Jerod Weinman, 13 August 2012
 * Revised 13 August 2014
 *
 * cu_bmalloc - Allocate memory for a bitmap on the CUDA device
 * hd_bmcopy - Copy the bitmap data from host bm_t to device memory
 * dh_bmcopy - Copy the bitmap data from device memory to host bm_t.
 * dd_bmcopy - Copy the bitmap data across device memory
 */

#include "cubitmap.h"


/* Allocate memory for a bm_t on the CUDA device */
bit* cu_bmalloc (int rows, int cols)
{
  
  bit* bits;
  
  hipError_t rc = hipMalloc( (void**)&bits,  cols * rows * sizeof(bit) );

  if (rc != hipSuccess) {
    fprintf(stderr,"Unable to allocate space on device for image: %s\n",
	    hipGetErrorString(rc));
    return NULL;
  }
  
  return bits;
}

/* Copy the data from host bm_t to device memory.
 * Assumes the buffers are the same size */
int hd_bmcopy( bit *dst, const bm_t *src)
{
  hipError_t rc = hipMemcpy( dst, src->bits,
			       sizeof(bit) * src->rows * src->cols,
			       hipMemcpyHostToDevice );
  
  if (rc != hipSuccess) 
  {
    fprintf(stderr,"Error copying image to device: %s\n",
	    hipGetErrorString(rc));
    return -1;
  }
  return 0;
}

/* Copy the data from device memory to host pbm_t.
 * Assumes the buffers are the same size */
int dh_bmcopy( bm_t *dst,  bit *src)
{
  hipError_t rc = hipMemcpy( dst->bits, src, 
			       dst->rows * dst->cols * sizeof(bit),
			       hipMemcpyDeviceToHost );
  
  if (rc != hipSuccess) {
    fprintf(stderr,"Error copying image to host: %s\n",
	    hipGetErrorString(rc));
    return -1;
  }
  return 0;
}

/* Copy the data across device memory .
 * Assumes the buffers are the same size */
int dd_bmcopy( bit *dst,  bit *src, int numRows, int numCols)
{
  hipError_t rc = hipMemcpy( dst, src,
			       numRows * numCols * sizeof(bit),
			       hipMemcpyDeviceToDevice );

  if (rc != hipSuccess) 
  {
    fprintf(stderr,"Error copying image across device: %s\n",
	    hipGetErrorString(rc));
    return -1;
  }
  return 0;
}