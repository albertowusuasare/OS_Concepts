#include "hip/hip_runtime.h"
/* Iterated Conditional Modes (ICM) binary image restoration
 *
 * Created by Jerod Weinman, 9 June 2008
 * Revised 15 August 2012 to include timing
 * Revised 13 August 2014 to use bitmap library and safely verify user input
 */
#include "cubitmap.h"
#include "bitmap.h"
#include <stdio.h>
#include <sys/time.h>
#include <string.h>

#define DEFAULT_ALPHA 2
#define DEFAULT_BETA 1
#define MAX_ITER 20
#define TILE_WIDTH 4

__device__ int alpha_d;
__device__ int beta_d;
__device__ int rows_d;
__device__ int cols_d;
__device__ int MAX_ITER_D;


/* icmupdate - Calculate one ICM update on an image
 *
 * Produces
 *  converged, an int
 *
 * Preconditions:
 *  img is the original binary image
 *  restImg is the current restored image result
 *  workImg is a buffer for storing the update
 *  img, restImg, and workImg are all the same size
 *
 * Postconditions:
 *  update is stored in workImg
 *  converged==1 indicates update yielded no change from restImg
 */


/*
 * Device update function
 */

__device__ int icmupdate(bit*img_d, bit * restImg_d, bit* workImg_d,int alpha,int beta,
                         int r,int c,int rows,int cols)
{
 int converged = 1; /* Flag indicating whether algorithm has converged */
  float cost[2]; /* cost array for both binary states at a pixel */
  bit *restBits = restImg_d;
  bit *workBits = workImg_d;
  //  int alpha = alpha_d;
  // int beta = beta_d;
 
  int index = r*cols + c;   /* Pixel linear index */

      cost[0] = 0;  /* Initialize costs to zero */
      cost[1] = 0;
      
      
      /* Local cost: for flipping pixel (r,c). This assigns the local
       * cost alpha to the opposite state of that at (r,c) */
      cost[1-img_d[index]] = alpha;
      
      /* Neighborhood cost: Adds beta to the cost for the opposite
       * value of each neighboring state. Note that the neighboring
       * state values used are from the most recent iteration of the
       * restored image. Also, ordered to maximize TLB hits. */
      if (r>0)
      {
	                cost[1-restBits[index-cols]]   += beta; /* North */
	if (c > 0)      cost[1-restBits[index-cols-1]] += beta; /* NorthWest */
	if (c < cols-1) cost[1-restBits[index-cols+1]] += beta; /* NorthEast */
      }
      if (c > 0)        cost[1-restBits[index-1]]      += beta; /* West */
      if (c < cols-1)   cost[1-restBits[index+1]]      += beta; /* East */
      if (r < rows-1)
      {
	                cost[1-restBits[index+cols]]   += beta;  /* South */
	if (c > 0)      cost[1-restBits[index+cols-1]] += beta; /* SouthWest */
	if (c < cols-1) cost[1-restBits[index+cols+1]] += beta; /* SouthEast */
      }

      /* Assign whichever state has lower cost to the intermediate
       * "working" restored image */
      workBits[index] = (cost[0] > cost[1]);
      
      /* If we still think we are converging, check whether the new
       * value (in workImg) differs from our previous restored
       * value. If they differ, we have not converged */
      if (converged && workBits[index]!=restBits[index])
	converged = 0;
    
  return converged;
}

/*
 * Kernel performing the icm Updates for each subset of pixels
 * img_d is the original image on the device
 * workImg_d is the work image on the device
 * restImg_d is the rest image on the device
 */

__global__ void icmKernel(bit *img_d, bit* restImg_d, bit *workImg_d, int alpha,int beta, 
                          int rows, int cols)
{

  /* Identify particular threads */
  int r = blockIdx.y * TILE_WIDTH + threadIdx.y; /* Pixel row */
  int c = blockIdx.x * TILE_WIDTH + threadIdx.x; /* Pixel col */


  int converged =0;
 int iter;      /* Iteration loop counter */
 
 for (iter=0 ; iter<MAX_ITER ; iter++)         /* Iterate update/restoration */
  {
    __syncthreads();
    converged = icmupdate (img_d,restImg_d, workImg_d,alpha,beta,r,c,rows,cols);     /* Update */
    __syncthreads();

    if( ( r*cols + c) ==  0)
      {
        printf("%d,%d\n",threadIdx.x,threadIdx.y);

          memcpy(restImg_d,workImg_d,sizeof(bit)* rows * cols);//copy working bits to restbits
      }
    // if (converged)
    //   break;         /* Nothing changed, so we are done and can exit the loop */
   // else          
      //   bmcopy(restImg_h,workImg_h);     /* All pixels updated; copy working buffer */
  }
}



/* runicm - Run the ICM algorthm with parameters alpha and beta on an image
 *
 * Preconditions:
 *  img is the original binary image
 *  restImg is an already allocated buffer for the restored image result
 *  img and restImg are the same size
 *
 * Postconditions:
 *  ICM is run on img until convergence or an iteration limit is reached
 *  Result is stored in restImg buffer
 */
__host__ void runicm(const bm_t *img_h, bm_t *restImg_h, double alpha, double beta)
{
  bm_t *workImg_h; /* An image buffer for storing intermediate results */
  int converged; /* Convergence flag for testing */
   int iter = MAX_ITER;      /* Iteration loop counter */

  int rows = img_h->rows;
  int cols = img_h->cols;
  
 workImg_h  = (bm_t*) malloc (sizeof(bm_t));          /* Allocate bitmap struct */
  
  if (workImg_h==NULL)
  {
    fprintf(stderr,"Unable to allocate work image");
    exit(EXIT_FAILURE);
  }

  if ( bmalloc(workImg_h, img_h->rows, img_h->cols) < 0 )    /* Allocate work image */
    exit(EXIT_FAILURE);
  
  if ( bmcopy(restImg_h, img_h) < 0 )              /* Copy original into restored */
    exit(EXIT_FAILURE);
  

  /* Allocate space on device */
  bit* img_d = cu_bmalloc(rows, cols);
  bit* restImg_d = cu_bmalloc(rows, cols);
  bit* workImg_d = cu_bmalloc(rows, cols);

  /* check if memory allocation was successful*/
  if (img_d==NULL || restImg_d==NULL || workImg_d==NULL) 
  {
    fprintf(stderr,"Error allocating device image memory\n");
    exit(EXIT_FAILURE);
  }

  /* Copy  image data to the device */

  
  if ( hd_bmcopy(img_d, img_h) < 0 ) /* Copy image to device */
  {
    fprintf(stderr,"Error copying original image to device\n");
    exit(EXIT_FAILURE);
  }

  if ( hd_bmcopy(restImg_d, restImg_h) < 0 ) /* Copy image to device */
  {
    fprintf(stderr,"Error copying rest image to device\n");
    exit(EXIT_FAILURE);
  }

  if ( hd_bmcopy(workImg_d, workImg_h) < 0 ) /* Copy image to device */
  {
    fprintf(stderr,"Error copying work image to device\n");
    exit(EXIT_FAILURE);
  }


  /* Allocate global constants on device*/

  hipError_t rc;

   // Copy value of alpha on host into alpha_d on device
    rc = hipMemcpyToSymbol(HIP_SYMBOL( alpha_d), &alpha, sizeof(int), 0, 
                             hipMemcpyHostToDevice);
 
    if (rc != hipSuccess)
        fprintf(stderr,"Unable to copy alpha value to device: %s", 
                hipGetErrorString(rc) );

      // Copy value of alpha on host into alpha_d on device
    rc = hipMemcpyToSymbol(HIP_SYMBOL( beta_d), &beta, sizeof(int), 0, 
                             hipMemcpyHostToDevice);
 
    if (rc != hipSuccess)
        fprintf(stderr,"Unable to copy beta value to device: %s", 
                hipGetErrorString(rc) );

        // Copy value of MAX_ITER on host into MAX_ITER_d on device
    rc = hipMemcpyToSymbol(HIP_SYMBOL( MAX_ITER_D), &iter, sizeof(int), 0, 
                             hipMemcpyHostToDevice);
 
    if (rc != hipSuccess)
        fprintf(stderr,"Unable to copy beta value to device: %s", 
                hipGetErrorString(rc) );

  /* Set up execution configuration */
  dim3 dimGrid( cols / TILE_WIDTH + 1, rows / TILE_WIDTH + 1 );
  dim3 dimBlock( TILE_WIDTH, TILE_WIDTH );
  
  /* Invoke Kernel*/
  icmKernel<<<dimGrid,dimBlock>>>(img_d,restImg_d,workImg_d,alpha,beta,rows,cols); 


  /*
   * Copy restored image from device to host
   */

  if (dh_bmcopy(restImg_h, restImg_d)<0) /* Copy inverted image to host */
  {
    fprintf(stderr,"Error copying restImg_d  to host");
    exit(EXIT_FAILURE);
  }

  /* free device memory */

  if (hipFree( img_d ) != hipSuccess ||   /* Free device memory */
      hipFree( restImg_d ) != hipSuccess ||
      hipFree( workImg_d )  != hipSuccess ) {
    fprintf(stderr,"Error releasing device memory\n");
  }


  // for (iter=0 ; iter<MAX_ITER ; iter++)         /* Iterate update/restoration */
  // {
  //   converged = icmupdate (img_h, restImg_h, workImg_h, alpha, beta);     /* Update */

  //   if (converged)
  //     break;         /* Nothing changed, so we are done and can exit the loop */
  //   else          
  //     bmcopy(restImg_h,workImg_h);     /* All pixels updated; copy working buffer */
  // }

  bmfree(workImg_h);                    /* Free our temporary work image buffer */
  free(workImg_h);
}

/* Run the ICM algorithm, write the result, and print the run time */
__host__ void run_time_icm(const char* origFile, const char* cleanFile,
		  double alpha, double beta)
{
  int result;                     /* Return value of gettimeofday for testing */
  struct timeval start,end, diff;          /* Clock and interval measurements */
  bm_t origImg, cleanImg;                          /* Input and output images */
  
  if (bmread(origFile, &origImg)<0)                       /* Read input image */
    exit (EXIT_FAILURE);
  
  if (bmalloc(&cleanImg, origImg.rows, origImg.cols) < 0) /* Allocate cleaned */
    exit (EXIT_FAILURE);

  if ( gettimeofday(&start, NULL) )                         /* Get start time */
  { 
    perror("Could not get start time");
    exit (EXIT_FAILURE);
  }

  runicm(&origImg, &cleanImg, alpha, beta);      /* Run restoration algorithm */

  if( gettimeofday(&end, NULL) )                              /* Get end time */
  {
    perror("Could not get end time");
    exit (EXIT_FAILURE);
  }
    
  timersub(&end, &start, &diff);                   /* Calculate time interval */
  printf("%u.%06u\n",diff.tv_sec,diff.tv_usec);           /* Print time (sec) */

  if (bmwrite(cleanFile, &cleanImg)<0)                /* Write restored image */
    exit (EXIT_FAILURE);
  
  bmfree(&cleanImg);                              /* Free our allocated image */
}

/* String to float conversion.
 * Preconditions: 
 *   All of str is the float
 *   str is a null-terminated character array
 *   cmd is a null-terminated character array
 * Postconditions: Prints an error message of the form 
 *   "cmd: name str must be a number" and exits the program with a failure when 
 *    the first precondition is violated. Otherwise returns the parsed number. */
__host__ float estrtof(char* str, char* cmd, const char* name )
{
  char* endPtr;             /* First unparseable character (for verification) */
  float num = strtof(str, &endPtr);

  if ( (endPtr-str) != strlen(str) ) /* Verify entire string was parsed */
  {
    fprintf(stderr,"%s: %s %s must be a number\n",cmd,name,str);
    exit (EXIT_FAILURE);
  }

  return num;
}

/* Verify and process command line arguments. 
 * Prints an error message and exits if any prove immediately problematic */
__host__ void processArguments(int argc, char* argv[],
		      char** origFile, char** cleanFile,
		      float *alpha, float *beta)
{
  if (argc<3 || argc>5)                     /* Verify optional argument count */
  {
    fprintf(stderr,"Usage: %s input output [alpha] [beta]\n", argv[0]);
    exit (EXIT_FAILURE);
  }
     
  *origFile = argv[1];
  *cleanFile = argv[2];

  if (argc>3)             /* Safely process optional alpha arguments */
    *alpha = estrtof(argv[3],argv[0],"alpha");
  else
    *alpha = DEFAULT_ALPHA;               /* No option given, take the default */
  
  if (argc>4)             /* Safely process optional alpha arguments */
    *beta = estrtof(argv[4],argv[0],"beta");
  else
    *beta = DEFAULT_BETA;                 /* No option given, take the default */
}
/* Main program: Process user input and calculate ICM result 
*  Usage: ./icm input output [alpha] [beta] */
int main(int argc, char* argv[])
{
  char *origFile, *cleanFile;                  /* Input and output file names */
  float alpha,beta;                               /* ICM algorithm parameters */
 
  processArguments(argc,argv,&origFile,&cleanFile,&alpha,&beta);
  run_time_icm(origFile, cleanFile, alpha, beta);    /* Time and write result */
  exit (EXIT_SUCCESS);                                        /* Exit cleanly */
}
